
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define M 20
#define N 20
#define THREADS_PER_BLOCK 5

// add function of matrix
__global__ void matmul(int *A, int *B, int *C){
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int col = blockIdx.y*blockDim.y+threadIdx.y;
    int value = 0;
    for(int k=0;k<N;k++){
        value = value+A[row*N+k]*B[k*N+col];
    }
    C[row*N+col] = value;
}

int main(){
    int *d_A,*d_B,*d_C;
    int size = M*N*sizeof (int);
    hipMalloc((void **)&d_A,size);
    hipMalloc((void **)&d_B,size);
    hipMalloc((void **)&d_C,size);

    // initialize the matrix
    int *A = (int *)malloc(size);
    int *B = (int *)malloc(size);
    int *C = (int *)malloc(size);
    for(int i=0;i<M*N;i++){
        A[i]=i;
        B[i]=i;
    }

    dim3 threadPerBlock(5, 5);
    dim3 blocks(M/threadPerBlock.x,N/threadPerBlock.y);

    // copy data into device
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

    matmul<<<blocks,threadPerBlock>>>(d_A,d_B,d_C);

    // copy back
    hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

    // print result:
    printf("The matrix add result is:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            printf("%d ", C[i * N + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}