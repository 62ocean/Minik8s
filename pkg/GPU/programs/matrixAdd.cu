
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define M 20
#define N 20
#define THREADS_PER_BLOCK 5

// add function of matrix
__global__ void matrix_add(int *A, int *B, int *C){
    C[threadIdx.x+blockIdx.x*THREADS_PER_BLOCK]=A[threadIdx.x+blockIdx.x*5]+B[threadIdx.x+blockIdx.x*5];
}

int main(){
    int *d_A,*d_B,*d_C;
    int size = M*N*sizeof (int);
    hipMalloc((void **)&d_A,size);
    hipMalloc((void **)&d_B,size);
    hipMalloc((void **)&d_C,size);

    // initialize the matrix
    int *A = (int *)malloc(size);
    int *B = (int *)malloc(size);
    int *C = (int *)malloc(size);
    for(int i=0;i<M*N;i++){
        A[i]=i;
        B[i]=i;
    }

    // copy data into device
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

    matrix_add<<<size/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_A,d_B,d_C);

    // copy back
    hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

    // print result:
    printf("The matrix add result is:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            printf("%d ", C[i * N + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}